
#include <hip/hip_runtime.h>

__global__ void
swap_reflect(float *A, int numElements)
{
    int i=blockIdx.x;
    int j=threadIdx.x;

    float temp;

    if ((i < numElements) && (j < numElements -1) && ((j)%2==0))
    {
        temp = A[i*numElements + j];
        A[i*numElements + j] = A[i*numElements + j + 1];
        A[i*numElements + j + 1] = temp;
        
    }
    __syncthreads();
    
    if((i < numElements) && (j < numElements) && (i>j) && (i!=j))
    {
        A[j*numElements + i] = A[i*numElements + j];
        //__syncthreads();
    }
}

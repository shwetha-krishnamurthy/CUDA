
#include <hip/hip_runtime.h>

__global__ void
convolution1d(float *A, float *B, float *M, int numElements)
{
    //int i=blockIdx.x;
    int i=threadIdx.x;
    int j, k;

    if (i < numElements)
    {
        for ( j = 0, k = -2; j < 4, k <=2; ++j, ++k)
        {
            if((i+k)>=0 && (i+k)<numElements)
            {
                B[i] += M[j]*A[i+k]; 
            }
        }
    }
   // __syncthreads();
}


__global__ void
convolution2d(float *A, float *B, float *M, int numElements)
{
    int i=blockIdx.x;
    int j=threadIdx.x;
    int k,l,m,n;

    if ((i < numElements) && (j < numElements))
    {
        for(k = 0, l = -1; k < 3, l <=1; k++, l++)
        {
            for(m = 0, n = -1; m < 3, n <=1; m++, n++)
            {
                if((i+l)>=0 && (i+l)<numElements && (j+n)>=0 && (j+n)<numElements)
                {
                    B[i*numElements + j] += M[k*3 + m]*A[(i+l)*numElements + (j+n)];
                }
            }
        } 
        __syncthreads();
    }
}
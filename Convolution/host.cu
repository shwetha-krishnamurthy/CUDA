#include "hip/hip_runtime.h"
#include "headers.h"
/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 5;
    int numElements2d = 5;
    int i, j;
    //printf("Enter matrix size: ");
    //scanf("%d", &numElements);
    size_t size = numElements * sizeof(float);
    size_t size2d = numElements2d * numElements2d * sizeof(float);
    //printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A, *h_B, *h_M;
    float *h_A2d, *h_B2d, *h_M2d;

    //for (i= 0; i<numElements; i++)
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_M = (float *)malloc(5*sizeof(float));

    h_A2d = (float *)malloc(size2d);
    h_B2d = (float *)malloc(size2d);
    h_M2d = (float *)malloc(9*sizeof(float));


    // Verify that allocations succeeded
    if (h_A == NULL) 
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    if (h_B == NULL) 
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    if (h_M == NULL) 
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    if (h_A2d == NULL) 
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    if (h_B2d == NULL) 
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    if (h_M2d == NULL) 
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    //Initialize vectors
 
    for (i = 0; i < numElements; ++i)
    {
        h_A[i] = ((float)rand()/(float)RAND_MAX);
        h_B[i] = 0;
        printf("%f, ", h_A[i]);
    }
    printf("\n");
    for (i = 0; i < 5; ++i)
    {
        h_M[i] = ((float)rand()/(float)RAND_MAX);
        printf("%f, ", h_M[i]);
    }
    printf("\n\n");


    //Initialize the host input vectors
    for (i = 0; i <  numElements; i++)
    {
      for (j = 0; j < numElements; j++)
      {
         h_A2d[i*numElements + j] = ((float)rand() / RAND_MAX);
         h_B2d[i*numElements + j] = 0;
         printf("%f ", h_A2d[i*numElements + j]); // Or *(*(arr+i)+j) = ++count
      }
      printf("\n");
    }

    printf("\n\n");
    
    //Initialize the host input vectors
    for (i = 0; i <  3; i++)
    {
      for (j = 0; j < 3; j++)
      {
         h_M2d[i*3 + j] = ((float)rand() / RAND_MAX);
         printf("%f ", h_M2d[i*3 + j]); // Or *(*(arr+i)+j) = ++count
      }
      printf("\n");
    }

    printf("\n\n");

    // Allocate the device input vector A
    float *d_A = NULL, *d_B = NULL, *d_M = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_M, 5*sizeof(float));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector M (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Allocate the device input vector A
    float *d_A2d = NULL, *d_B2d = NULL, *d_M2d = NULL;
    err = hipMalloc((void **)&d_A2d, size2d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_B2d, size2d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_M2d, 9*sizeof(float));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector M (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_M, h_M, 5*sizeof(float), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector M from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A2d, h_A2d, size2d, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_B2d, h_B2d, size2d, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_M2d, h_M2d, 9*sizeof(float), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector M from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = numElements;
    int blocksPerGrid = numElements;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    convolution1d<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_M, numElements);
    err = hipGetLastError();

    int threadsPerBlock2d = numElements2d;
    int blocksPerGrid2d = numElements2d;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid2d, threadsPerBlock2d);
    convolution2d<<<blocksPerGrid2d, threadsPerBlock2d>>>(d_A2d, d_B2d, d_M2d, numElements2d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch swap kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_B2d, d_B2d, size2d, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    // for (int i = 0; i < numElements; ++i)
    // {
    //     if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
    //     {
    //         fprintf(stderr, "Result verification failed at element %d!\n", i);
    //         exit(EXIT_FAILURE);
    //     }
    // }

    // for(i = 0; i<numElements; i++)
    // {
    // 	for (j = 0; j<numElements; j++)
    // 	{
    // 		printf("%f ", h_A[i*numElements + j]);
    // 	}
    // 	printf("\n");
    // }

    for (i = 0; i < numElements; ++i)
    {
        printf("%f, ", h_B[i]);
    }

    printf("\n");


    for(i = 0; i<numElements; i++)
    {
        for (j = 0; j<numElements; j++)
        {
            printf("%f ", h_B2d[i*numElements + j]);
        }
        printf("\n");
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_A2d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B2d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_M);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_M2d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // err = hipFree(d_C);

    // if (err != hipSuccess)
    // {
    //     fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_A2d);
    free(h_B2d);
    free(h_M2d);
    free(h_M2d);
    // free(h_C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

